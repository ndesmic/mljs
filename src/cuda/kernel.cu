#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "kernel.h"
#include "stdio.h"

__device__ void insertAtIndex(const int sourcePtr, const int length, const int destinationPtr, const int indexToInsert, const int value, char *mem)
{
    int sourceIndex = 0;
    int destinationIndex = 0;

    while (destinationIndex < length + 1)
    {
        if (destinationIndex != indexToInsert)
        {
            mem[destinationPtr + destinationIndex] = mem[sourcePtr + sourceIndex];
            sourceIndex++;
            destinationIndex++;
        }
        else
        {
            mem[destinationPtr + destinationIndex] = value;
            destinationIndex++;
        }
    }
}

__device__ void removeAtIndex(const int sourcePtr, const int length, const int destinationPtr, const int indexToRemove, char *mem)
{
    int sourceIndex = 0;
    int destinationIndex = 0;

    while (destinationIndex < length - 1)
    {
        if (sourceIndex != indexToRemove)
        {
            mem[destinationPtr + destinationIndex] = mem[sourcePtr + sourceIndex];
            sourceIndex++;
            destinationIndex++;
        }
        else
        {
            sourceIndex++;
        }
    }
}

__device__ void getDimensionalIndices(int flatIndex, const int shapePtr, const int shapeSize, const int destinationPtr, char *mem)
{
    int currentIndex = flatIndex;

    for (int i = 0; i < shapeSize; i++)
    {
        mem[destinationPtr + i] = currentIndex % mem[shapePtr + i];
        currentIndex = currentIndex / mem[shapePtr + i];
    }
}

__device__ int getFlatIndex(const int dimensionalIndexPtr, const int shapePtr, const int shapeSize, char *mem)
{
    int index = 0;
    for (int i = 0; i < shapeSize; i++)
    {
        index *= mem[shapePtr + shapeSize - 1 - i];
        index += mem[dimensionalIndexPtr + shapeSize - 1 - i];
    }
    return index;
}

__global__ void addKernel(const int size, const float *valuesA, const float *valuesB, float *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        output[idx] = valuesA[idx] + valuesB[idx];
    }
}
__global__ void addBackpropKernel(const int size, float *gradA, float *gradB, const float *gradResult)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        gradA[idx] += gradResult[idx];
        gradB[idx] += gradResult[idx];
    }
}
__global__ void subKernel(const int size, const float *valuesA, const float *valuesB, float *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        output[idx] = valuesA[idx] - valuesB[idx];
    }
}
__global__ void subBackpropKernel(const int size, float *gradA, float *gradB, const float *gradResult)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        gradA[idx] += gradResult[idx];
        gradB[idx] += -1 * gradResult[idx];
    }
}
__global__ void mulKernel(const int size, const float *valuesA, const float *valuesB, float *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        output[idx] = valuesA[idx] * valuesB[idx];
    }
}
__global__ void mulBackpropKernel(const int size, float *gradA, float *gradB, const float *gradResult, const float *valuesA, const float *valuesB)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        gradA[idx] += valuesB[idx] * gradResult[idx];
        gradB[idx] += valuesA[idx] * gradResult[idx];
    }
}
__global__ void divKernel(const int size, const float *valuesA, const float *valuesB, float *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        output[idx] = valuesA[idx] / valuesB[idx];
    }
}
__global__ void divBackpropKernel(const int size, float *gradA, float *gradB, const float *gradResult, const float *valuesA, const float *valuesB)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        gradA[idx] += 1 / valuesB[idx] * gradResult[idx];
        gradB[idx] += -1 * valuesA[idx] / pow(valuesB[idx], 2) * gradResult[idx];
    }
}
__global__ void powKernel(const int size, const float *valuesA, const float *valuesB, float *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        output[idx] = pow(valuesA[idx], valuesB[idx]);
    }
}
__global__ void powBackpropKernel(const int size, float *gradA, float *gradB, const float *gradResult, const float *valuesA, const float *valuesB)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        gradA[idx] += valuesB[idx] * pow(valuesA[idx], valuesB[idx] - 1) * gradResult[idx];
        gradB[idx] += log(valuesA[idx]) * pow(valuesA[idx], valuesB[idx]) * gradResult[idx];
    }
}
__global__ void negKernel(const int size, const float *values, float *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        output[idx] = -values[idx];
    }
}
__global__ void negBackpropKernel(const int size, float *grad, const float *gradResult)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        grad[idx] += -1 * gradResult[idx];
    }
}
__global__ void expKernel(const int size, const float *values, float *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        output[idx] = exp(values[idx]);
    }
}
__global__ void expBackpropKernel(const int size, float *grad, const float *gradResult, const float *values)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        grad[idx] += exp(values[idx]) * gradResult[idx];
    }
}
__global__ void tanhKernel(const int size, const float *values, float *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        output[idx] = tanh(values[idx]);
    }
}
__global__ void tanhBackpropKernel(const int size, float *grad, const float *gradResult, const float *values)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        grad[idx] += (1 - pow(tanh(values[idx]), 2)) * gradResult[idx];
    }
}
__global__ void sumKernel(const int *shape, const int shapeSize, const int dimToReduce, const float *values, float *output, char *mem)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int newSize = 1;
    for (int j = 0; j < shapeSize; j++)
    {
        if (j != dimToReduce)
        {
            newSize *= shape[j];
        }
    }
    if (idx < newSize)
    {
        size_t basePtr = idx * ((shapeSize * 4) - 2); //manual calc :/
        size_t memPtr = basePtr;
        size_t shapePtr = basePtr;

        for (int i = 0; i < shapeSize; i++)
        {
            mem[memPtr] = shape[i];
            memPtr++;
        }

        size_t newShapePtr = memPtr;
        removeAtIndex(shapePtr, shapeSize, newShapePtr, dimToReduce, mem);
        memPtr += shapeSize - 1;

        size_t partialDimIndexPtr = memPtr;
        getDimensionalIndices(idx, newShapePtr, shapeSize - 1, partialDimIndexPtr, mem);
        memPtr += shapeSize - 1;

        for (int i = 0; i < shape[dimToReduce]; i++)
        {
            size_t dimIndexPtr = memPtr;
            insertAtIndex(partialDimIndexPtr, shapeSize - 1, dimIndexPtr, dimToReduce, i, mem);

            size_t flatIdx = getFlatIndex(dimIndexPtr, shapePtr, shapeSize, mem);
            output[idx] += values[flatIdx];
        }
    }
}

__global__ void sumBackpropKernel(const int *shape, const int shapeSize, const int dimToReduce, float *grad, const float *gradResult, char* mem)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int inSize = 1;
    for (int j = 0; j < shapeSize; j++)
    {
        inSize *= shape[j];
    }

    if (idx < inSize)
    {
        size_t basePtr = idx * ((shapeSize * 4) - 2); //manual calc :/
        size_t memPtr = basePtr;
        size_t shapePtr = basePtr;

        for(int i = 0; i < shapeSize; i++){
            mem[memPtr] = shape[i];
            memPtr++;
        }

        size_t outShapePtr = memPtr;
        removeAtIndex(shapePtr, shapeSize, outShapePtr, dimToReduce, mem);
        memPtr += shapeSize - 1;

        size_t inDimIndexPtr = memPtr;
        getDimensionalIndices(idx, shapePtr, shapeSize, inDimIndexPtr, mem);
        memPtr += shapeSize;

        size_t outDimIndexPtr = memPtr;
        removeAtIndex(inDimIndexPtr, shapeSize, outDimIndexPtr, dimToReduce, mem);
        memPtr += shapeSize - 1;


        size_t outputFlatIdx = getFlatIndex(outDimIndexPtr, outShapePtr, shapeSize - 1, mem);

        grad[idx] += gradResult[outputFlatIdx];
    }
}

// Binary Ops

float *add_op(const int size, const float *valuesA, const float *valuesB)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *output = new float[size];

    float *valuesA_d;
    float *valuesB_d;
    float *output_d;

    hipMalloc((void **)&valuesA_d, size * sizeof(float));
    hipMalloc((void **)&valuesB_d, size * sizeof(float));
    hipMalloc((void **)&output_d, size * sizeof(float));

    hipMemcpy(valuesA_d, valuesA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesB_d, valuesB, size * sizeof(float), hipMemcpyHostToDevice);

    addKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, valuesA_d, valuesB_d, output_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(output, output_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(valuesA_d);
    hipFree(valuesB_d);
    hipFree(output_d);

    return output;
}

void addBackprop_op(const int size, float *gradA, float *gradB, const float *gradResult)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *gradA_d;
    float *gradB_d;
    float *gradResult_d;

    hipMalloc((void **)&gradA_d, size * sizeof(float));
    hipMalloc((void **)&gradB_d, size * sizeof(float));
    hipMalloc((void **)&gradResult_d, size * sizeof(float));

    hipMemcpy(gradA_d, gradA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradB_d, gradB, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradResult_d, gradResult, size * sizeof(float), hipMemcpyHostToDevice);

    addBackpropKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradResult_d);

    if (gradA == gradB)
    { // if same reference then we need to add them
        float *gradUnified_d;
        hipMalloc((void **)&gradUnified_d, size * sizeof(float));
        addKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradUnified_d);

        hipMemcpy(gradA, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(gradUnified_d);
    }
    else
    {
        hipMemcpy(gradA, gradA_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradB_d, size * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipFree(gradA_d);
    hipFree(gradB_d);
    hipFree(gradResult_d);
}

float *sub_op(const int size, const float *valuesA, const float *valuesB)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *output = new float[size];

    float *valuesA_d;
    float *valuesB_d;
    float *output_d;

    hipMalloc((void **)&valuesA_d, size * sizeof(float));
    hipMalloc((void **)&valuesB_d, size * sizeof(float));
    hipMalloc((void **)&output_d, size * sizeof(float));

    hipMemcpy(valuesA_d, valuesA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesB_d, valuesB, size * sizeof(float), hipMemcpyHostToDevice);

    subKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, valuesA_d, valuesB_d, output_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(output, output_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(valuesA_d);
    hipFree(valuesB_d);
    hipFree(output_d);

    return output;
}

void subBackprop_op(const int size, float *gradA, float *gradB, const float *gradResult)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *gradA_d;
    float *gradB_d;
    float *gradResult_d;

    hipMalloc((void **)&gradA_d, size * sizeof(float));
    hipMalloc((void **)&gradB_d, size * sizeof(float));
    hipMalloc((void **)&gradResult_d, size * sizeof(float));

    hipMemcpy(gradA_d, gradA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradB_d, gradB, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradResult_d, gradResult, size * sizeof(float), hipMemcpyHostToDevice);

    subBackpropKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradResult_d);

    if (gradA == gradB)
    { // if same reference then we need to add them
        float *gradUnified_d;
        hipMalloc((void **)&gradUnified_d, size * sizeof(float));
        addKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradUnified_d);

        hipMemcpy(gradA, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(gradUnified_d);
    }
    else
    {
        hipMemcpy(gradA, gradA_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradB_d, size * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipFree(gradA_d);
    hipFree(gradB_d);
    hipFree(gradResult_d);
}

float *mul_op(const int size, const float *valuesA, const float *valuesB)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *output = new float[size];

    float *valuesA_d;
    float *valuesB_d;
    float *output_d;

    hipMalloc((void **)&valuesA_d, size * sizeof(float));
    hipMalloc((void **)&valuesB_d, size * sizeof(float));
    hipMalloc((void **)&output_d, size * sizeof(float));

    hipMemcpy(valuesA_d, valuesA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesB_d, valuesB, size * sizeof(float), hipMemcpyHostToDevice);

    mulKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, valuesA_d, valuesB_d, output_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(output, output_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(valuesA_d);
    hipFree(valuesB_d);
    hipFree(output_d);

    return output;
}

void mulBackprop_op(const int size, float *gradA, float *gradB, const float *gradResult, const float *valuesA, const float *valuesB)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *gradA_d;
    float *gradB_d;
    float *gradResult_d;
    float *valuesA_d;
    float *valuesB_d;

    hipMalloc((void **)&gradA_d, size * sizeof(float));
    hipMalloc((void **)&gradB_d, size * sizeof(float));
    hipMalloc((void **)&gradResult_d, size * sizeof(float));
    hipMalloc((void **)&valuesA_d, size * sizeof(float));
    hipMalloc((void **)&valuesB_d, size * sizeof(float));

    hipMemcpy(gradA_d, gradA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradB_d, gradB, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradResult_d, gradResult, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesA_d, valuesA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesB_d, valuesB, size * sizeof(float), hipMemcpyHostToDevice);

    mulBackpropKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradResult_d, valuesA_d, valuesB_d);

    if (gradA == gradB)
    { // if same reference then we need to add them
        float *gradUnified_d;
        hipMalloc((void **)&gradUnified_d, size * sizeof(float));
        addKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradUnified_d);

        hipMemcpy(gradA, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(gradUnified_d);
    }
    else
    {
        hipMemcpy(gradA, gradA_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradB_d, size * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipFree(gradA_d);
    hipFree(gradB_d);
    hipFree(gradResult_d);
    hipFree(valuesA_d);
    hipFree(valuesB_d);
}

float *div_op(const int size, const float *valuesA, const float *valuesB)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *output = new float[size];

    float *valuesA_d;
    float *valuesB_d;
    float *output_d;

    hipMalloc((void **)&valuesA_d, size * sizeof(float));
    hipMalloc((void **)&valuesB_d, size * sizeof(float));
    hipMalloc((void **)&output_d, size * sizeof(float));

    hipMemcpy(valuesA_d, valuesA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesB_d, valuesB, size * sizeof(float), hipMemcpyHostToDevice);

    divKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, valuesA_d, valuesB_d, output_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(output, output_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(valuesA_d);
    hipFree(valuesB_d);
    hipFree(output_d);

    return output;
}

void divBackprop_op(const int size, float *gradA, float *gradB, const float *gradResult, const float *valuesA, const float *valuesB)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *gradA_d;
    float *gradB_d;
    float *gradResult_d;
    float *valuesA_d;
    float *valuesB_d;

    hipMalloc((void **)&gradA_d, size * sizeof(float));
    hipMalloc((void **)&gradB_d, size * sizeof(float));
    hipMalloc((void **)&gradResult_d, size * sizeof(float));
    hipMalloc((void **)&valuesA_d, size * sizeof(float));
    hipMalloc((void **)&valuesB_d, size * sizeof(float));

    hipMemcpy(gradA_d, gradA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradB_d, gradB, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradResult_d, gradResult, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesA_d, valuesA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesB_d, valuesB, size * sizeof(float), hipMemcpyHostToDevice);

    divBackpropKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradResult_d, valuesA_d, valuesB_d);

    if (gradA == gradB)
    { // if same reference then we need to add them

        float *ggA = (float *)malloc(size * sizeof(float));
        float *ggB = (float *)malloc(size * sizeof(float));
        hipMemcpy(ggA, gradA_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(ggB, gradB_d, size * sizeof(float), hipMemcpyDeviceToHost);

        float *gradUnified_d;
        hipMalloc((void **)&gradUnified_d, size * sizeof(float));
        addKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradUnified_d);

        hipMemcpy(gradA, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(gradUnified_d);
    }
    else
    {
        hipMemcpy(gradA, gradA_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradB_d, size * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipFree(gradA_d);
    hipFree(gradB_d);
    hipFree(gradResult_d);
    hipFree(valuesA_d);
    hipFree(valuesB_d);
}

float *pow_op(const int size, const float *valuesA, const float *valuesB)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *output = new float[size];

    float *valuesA_d;
    float *valuesB_d;
    float *output_d;

    hipMalloc((void **)&valuesA_d, size * sizeof(float));
    hipMalloc((void **)&valuesB_d, size * sizeof(float));
    hipMalloc((void **)&output_d, size * sizeof(float));

    hipMemcpy(valuesA_d, valuesA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesB_d, valuesB, size * sizeof(float), hipMemcpyHostToDevice);

    powKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, valuesA_d, valuesB_d, output_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(output, output_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(valuesA_d);
    hipFree(valuesB_d);
    hipFree(output_d);

    return output;
}

void powBackprop_op(const int size, float *gradA, float *gradB, const float *gradResult, const float *valuesA, const float *valuesB)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *gradA_d;
    float *gradB_d;
    float *gradResult_d;
    float *valuesA_d;
    float *valuesB_d;

    hipMalloc((void **)&gradA_d, size * sizeof(float));
    hipMalloc((void **)&gradB_d, size * sizeof(float));
    hipMalloc((void **)&gradResult_d, size * sizeof(float));
    hipMalloc((void **)&valuesA_d, size * sizeof(float));
    hipMalloc((void **)&valuesB_d, size * sizeof(float));

    hipMemcpy(gradA_d, gradA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradB_d, gradB, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradResult_d, gradResult, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesA_d, valuesA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesB_d, valuesB, size * sizeof(float), hipMemcpyHostToDevice);

    powBackpropKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradResult_d, valuesA_d, valuesB_d);

    if (gradA == gradB)
    { // if same reference then we need to add them
        float *gradUnified_d;
        hipMalloc((void **)&gradUnified_d, size * sizeof(float));
        addKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradUnified_d);

        hipMemcpy(gradA, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(gradUnified_d);
    }
    else
    {
        hipMemcpy(gradA, gradA_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradB_d, size * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipFree(gradA_d);
    hipFree(gradB_d);
    hipFree(gradResult_d);
    hipFree(valuesA_d);
    hipFree(valuesB_d);
}

// Unary Ops
float *neg_op(const int size, const float *values)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *output = new float[size];

    float *values_d;
    float *output_d;

    hipMalloc((void **)&values_d, size * sizeof(float));
    hipMalloc((void **)&output_d, size * sizeof(float));

    hipMemcpy(values_d, values, size * sizeof(float), hipMemcpyHostToDevice);

    negKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, values_d, output_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(output, output_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(values_d);
    hipFree(output_d);

    return output;
}
void negBackprop_op(const int size, float *grad, const float *gradResult)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *grad_d;
    float *gradResult_d;

    hipMalloc((void **)&grad_d, size * sizeof(float));
    hipMalloc((void **)&gradResult_d, size * sizeof(float));

    hipMemcpy(grad_d, grad, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradResult_d, gradResult, size * sizeof(float), hipMemcpyHostToDevice);

    negBackpropKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, grad_d, gradResult_d);

    hipMemcpy(grad, grad_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipFree(grad_d);
    hipFree(gradResult_d);
}
float *exp_op(const int size, const float *values)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *output = new float[size];

    float *values_d;
    float *output_d;

    hipMalloc((void **)&values_d, size * sizeof(float));
    hipMalloc((void **)&output_d, size * sizeof(float));

    hipMemcpy(values_d, values, size * sizeof(float), hipMemcpyHostToDevice);

    expKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, values_d, output_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(output, output_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(values_d);
    hipFree(output_d);

    return output;
}
void expBackprop_op(const int size, float *grad, const float *gradResult, const float *values)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *grad_d;
    float *gradResult_d;
    float *values_d;

    hipMalloc((void **)&grad_d, size * sizeof(float));
    hipMalloc((void **)&gradResult_d, size * sizeof(float));
    hipMalloc((void **)&values_d, size * sizeof(float));

    hipMemcpy(grad_d, grad, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradResult_d, gradResult, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(values_d, values, size * sizeof(float), hipMemcpyHostToDevice);

    expBackpropKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, grad_d, gradResult_d, values_d);

    hipMemcpy(grad, grad_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipFree(grad_d);
    hipFree(gradResult_d);
    hipFree(values_d);
}
float *tanh_op(const int size, const float *values)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *output = new float[size];

    float *values_d;
    float *output_d;

    hipMalloc((void **)&values_d, size * sizeof(float));
    hipMalloc((void **)&output_d, size * sizeof(float));

    hipMemcpy(values_d, values, size * sizeof(float), hipMemcpyHostToDevice);

    tanhKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, values_d, output_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(output, output_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(values_d);
    hipFree(output_d);

    return output;
}
void tanhBackprop_op(const int size, float *grad, const float *gradResult, const float *values)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *grad_d;
    float *gradResult_d;
    float *values_d;

    hipMalloc((void **)&grad_d, size * sizeof(float));
    hipMalloc((void **)&gradResult_d, size * sizeof(float));
    hipMalloc((void **)&values_d, size * sizeof(float));

    hipMemcpy(grad_d, grad, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradResult_d, gradResult, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(values_d, values, size * sizeof(float), hipMemcpyHostToDevice);

    tanhBackpropKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, grad_d, gradResult_d, values_d);

    hipMemcpy(grad, grad_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipFree(grad_d);
    hipFree(gradResult_d);
    hipFree(values_d);
}

// Reduction Ops

float *sum_op(const int *shape, const int shapeSize, const int dimToReduce, const float *values)
{
    int BLOCK_SIZE = 32;

    int size = 1;
    for (int i = 0; i < shapeSize; i++)
    {
        size *= shape[i];
    }
    int outSize = size / shape[dimToReduce];

    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *output = new float[outSize];

    float *values_d;
    float *output_d;
    int *shape_d;
    char*mem_d;

    hipMalloc((void **)&values_d, size * sizeof(float));
    hipMalloc((void **)&output_d, outSize * sizeof(float));
    hipMalloc((void **)&shape_d, shapeSize * sizeof(int));

    // memory areana
    int threadMemSize = ((shapeSize * 4) - 2) * 4;
    hipMalloc((void **)&mem_d, threadMemSize * outSize);

    hipMemcpy(values_d, values, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(shape_d, shape, shapeSize * sizeof(int), hipMemcpyHostToDevice);

    sumKernel<<<GRID_SIZE, BLOCK_SIZE>>>(shape_d, shapeSize, dimToReduce, values_d, output_d, mem_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(output, output_d, outSize * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(values_d);
    hipFree(output_d);
    hipFree(shape_d);
    hipFree(mem_d);

    return output;
}

void sumBackprop_op(const int *shape, const int shapeSize, const int dimToReduce, float *grad, const float *gradResult)
{
    int BLOCK_SIZE = 32;

    int size = 1;
    for (int i = 0; i < shapeSize; i++)
    {
        size *= shape[i];
    }
    int outSize = size / shape[dimToReduce];

    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *grad_d;
    float *gradResult_d;
    int *shape_d;
    char *mem_d;

    hipMalloc((void **)&grad_d, size * sizeof(float));
    hipMalloc((void **)&gradResult_d, size * sizeof(float));
    hipMalloc((void **)&shape_d, shapeSize * sizeof(int));

    // memory areana
    int threadMemSize = ((shapeSize * 4) - 2) * 4;
    hipMalloc((void **)&mem_d, threadMemSize * outSize);

    hipMemcpy(grad_d, grad, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradResult_d, gradResult, outSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(shape_d, shape, shapeSize * sizeof(int), hipMemcpyHostToDevice);

    sumBackpropKernel<<<GRID_SIZE, BLOCK_SIZE>>>(shape_d, shapeSize, dimToReduce, grad_d, gradResult_d, mem_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(grad, grad_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(grad_d);
    hipFree(gradResult_d);
    hipFree(shape_d);
    hipFree(mem_d);
}