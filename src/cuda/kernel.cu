#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "kernel.h"
#include "stdio.h"

__device__ int* insertAtIndex(const int* array, const int length, const int index, const int value){
    int* outArray = (int *)malloc((length + 1) * sizeof(int));
    int sourceIndex = 0;
    int destinationIndex = 0;
    while(destinationIndex < length + 1){
        if(destinationIndex != index){
            outArray[destinationIndex] = array[sourceIndex];
            sourceIndex++;
            destinationIndex++;
        } else {
            outArray[destinationIndex] = value;
            destinationIndex++;
        }
    }
    return outArray;
}
__device__ int *removeAtIndex(const int *array, const int length, const int index)
{
    int *outArray = (int *)malloc((length - 1) * sizeof(int));
    int sourceIndex = 0;
    int destinationIndex = 0;
    while (destinationIndex < length - 1)
    {
        if (sourceIndex != index)
        {
            outArray[destinationIndex] = array[sourceIndex];
            sourceIndex++;
            destinationIndex++;
        }
        else
        {
            sourceIndex++;
        }
    }
    return outArray;
}
//shape is column major
__device__ int* getDimensionalIndex(int flatIndex, const int shape[], const int shapeSize)
{
    int* indices = (int*)malloc(sizeof(shape));
    for(int i = 0; i < shapeSize; i++){
        indices[i] = flatIndex % shape[i];
        flatIndex = flatIndex / shape[i];
    }
    return indices;
}

//shape and dimensionalIndex are column major
__device__ int getFlatIndex(const int dimensionalIndex[], const int shape[], const int shapeSize)
{
    int index = 0;
    for (int i = 0; i < shapeSize; i++)
    {
        index *= shape[shapeSize - 1 - i];
        index += dimensionalIndex[shapeSize - 1 - i];
    }
    return index;
}
__global__ void addKernel(const int size, const float* valuesA, const float* valuesB, float* output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {        
        output[idx] = valuesA[idx] + valuesB[idx];
    }
}
__global__ void addBackpropKernel(const int size, float *gradA, float *gradB, const float *gradResult)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        gradA[idx] += gradResult[idx];
        gradB[idx] += gradResult[idx];
    }
}
__global__ void subKernel(const int size, const float *valuesA, const float *valuesB, float *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        output[idx] = valuesA[idx] - valuesB[idx];
    }
}
__global__ void subBackpropKernel(const int size, float *gradA, float *gradB, const float *gradResult)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        gradA[idx] += gradResult[idx];
        gradB[idx] += -1 * gradResult[idx];
    }
}
__global__ void mulKernel(const int size, const float *valuesA, const float *valuesB, float *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        output[idx] = valuesA[idx] * valuesB[idx];
    }
}
__global__ void mulBackpropKernel(const int size, float *gradA, float *gradB, const float *gradResult, const float *valuesA, const float *valuesB )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        gradA[idx] += valuesB[idx] * gradResult[idx];
        gradB[idx] += valuesA[idx] * gradResult[idx];
    }
}
__global__ void divKernel(const int size, const float *valuesA, const float *valuesB, float *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        output[idx] = valuesA[idx] / valuesB[idx];
    }
}
__global__ void divBackpropKernel(const int size, float *gradA, float *gradB, const float *gradResult, const float *valuesA, const float *valuesB)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        gradA[idx] += 1 / valuesB[idx] * gradResult[idx];
        gradB[idx] += -1 * valuesA[idx] / pow(valuesB[idx], 2) * gradResult[idx];
    }
}
__global__ void powKernel(const int size, const float *valuesA, const float *valuesB, float *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        output[idx] = pow(valuesA[idx], valuesB[idx]);
    }
}
__global__ void powBackpropKernel(const int size, float *gradA, float *gradB, const float *gradResult, const float *valuesA, const float *valuesB)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        gradA[idx] += valuesB[idx] * pow(valuesA[idx], valuesB[idx] - 1) * gradResult[idx];
        gradB[idx] += log(valuesA[idx]) * pow(valuesA[idx], valuesB[idx]) * gradResult[idx];
    }
}
__global__ void negKernel(const int size, const float *values, float *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        output[idx] = -values[idx];
    }
}
__global__ void negBackpropKernel(const int size, float *grad, const float *gradResult)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        grad[idx] += -1 * gradResult[idx];
    }
}
__global__ void expKernel(const int size, const float *values, float *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        output[idx] = exp(values[idx]);
    }
}
__global__ void expBackpropKernel(const int size, float *grad, const float *gradResult, const float *values)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        grad[idx] += exp(values[idx]) * gradResult[idx];
    }
}
__global__ void tanhKernel(const int size, const float *values, float *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        output[idx] = tanh(values[idx]);
    }
}
__global__ void tanhBackpropKernel(const int size, float *grad, const float *gradResult, const float *values)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        grad[idx] += (1 - pow(tanh(values[idx]), 2)) * gradResult[idx];
    }
}
__global__ void sumKernel(const int* shape, const int shapeSize, const int dimToReduce, const float* values, float* output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int newSize = 1;
    for (int j = 0; j < shapeSize; j++)
    {
        if (j != dimToReduce)
        {
            newSize *= shape[j];
        }
    }
    if (idx < newSize)
    {
        int *newShape = removeAtIndex(shape, shapeSize, dimToReduce);
        int *partialDimIndex = getDimensionalIndex(idx, shape, shapeSize - 1);

        for(int i = 0; i < shape[dimToReduce]; i++){
            int* dimIndex = insertAtIndex(partialDimIndex, shapeSize - 1, dimToReduce, i);
            int flatIdx = getFlatIndex(dimIndex, shape, shapeSize);
            output[idx] += values[getFlatIndex(dimIndex, shape, shapeSize)];
            free(dimIndex);
        }

        free(newShape);
        free(partialDimIndex);
    }
}

__global__ void sumBackpropKernel(const int *shape, const int shapeSize, const int dimToReduce, float *grad, const float *gradResult)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int inSize = 1;
    for (int j = 0; j < shapeSize; j++)
    {
        inSize *= shape[j];
    }

    if (idx < inSize)
    {
        int *outShape = removeAtIndex(shape, shapeSize, dimToReduce);
        int *inDimIndex = getDimensionalIndex(idx, shape, shapeSize);
        int *outDimIndex = removeAtIndex(inDimIndex, shapeSize, dimToReduce);
        int outputFlatIdx = getFlatIndex(outDimIndex, outShape, shapeSize - 1);

        grad[idx] += gradResult[outputFlatIdx];

        free(outDimIndex);
        free(outShape);
        free(inDimIndex);
    }
}

//Binary Ops

float *add_op(const int size, const float *valuesA, const float *valuesB)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *output = new float[size];

    float *valuesA_d;
    float *valuesB_d;
    float *output_d;

    hipMalloc((void **)&valuesA_d, size * sizeof(float));
    hipMalloc((void **)&valuesB_d, size * sizeof(float));
    hipMalloc((void **)&output_d, size * sizeof(float));

    hipMemcpy(valuesA_d, valuesA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesB_d, valuesB, size * sizeof(float), hipMemcpyHostToDevice);

    addKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, valuesA_d, valuesB_d, output_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(output, output_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(valuesA_d);
    hipFree(valuesB_d);
    hipFree(output_d);

    return output;
}

void addBackprop_op(const int size, float *gradA, float *gradB, const float *gradResult)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *gradA_d;
    float *gradB_d;
    float *gradResult_d;

    hipMalloc((void **)&gradA_d, size * sizeof(float));
    hipMalloc((void **)&gradB_d, size * sizeof(float));
    hipMalloc((void **)&gradResult_d, size * sizeof(float));

    hipMemcpy(gradA_d, gradA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradB_d, gradB, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradResult_d, gradResult, size * sizeof(float), hipMemcpyHostToDevice);

    addBackpropKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradResult_d);

    if (gradA == gradB)
    { // if same reference then we need to add them
        float *gradUnified_d;
        hipMalloc((void **)&gradUnified_d, size * sizeof(float));
        addKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradUnified_d);

        hipMemcpy(gradA, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(gradUnified_d);
    }
    else
    {
        hipMemcpy(gradA, gradA_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradB_d, size * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipFree(gradA_d);
    hipFree(gradB_d);
    hipFree(gradResult_d);
}

float *sub_op(const int size, const float *valuesA, const float *valuesB)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *output = new float[size];

    float *valuesA_d;
    float *valuesB_d;
    float *output_d;

    hipMalloc((void **)&valuesA_d, size * sizeof(float));
    hipMalloc((void **)&valuesB_d, size * sizeof(float));
    hipMalloc((void **)&output_d, size * sizeof(float));

    hipMemcpy(valuesA_d, valuesA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesB_d, valuesB, size * sizeof(float), hipMemcpyHostToDevice);

    subKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, valuesA_d, valuesB_d, output_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(output, output_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(valuesA_d);
    hipFree(valuesB_d);
    hipFree(output_d);

    return output;
}

void subBackprop_op(const int size, float *gradA, float *gradB, const float *gradResult)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *gradA_d;
    float *gradB_d;
    float *gradResult_d;

    hipMalloc((void **)&gradA_d, size * sizeof(float));
    hipMalloc((void **)&gradB_d, size * sizeof(float));
    hipMalloc((void **)&gradResult_d, size * sizeof(float));

    hipMemcpy(gradA_d, gradA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradB_d, gradB, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradResult_d, gradResult, size * sizeof(float), hipMemcpyHostToDevice);

    subBackpropKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradResult_d);

    if (gradA == gradB)
    { // if same reference then we need to add them
        float *gradUnified_d;
        hipMalloc((void **)&gradUnified_d, size * sizeof(float));
        addKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradUnified_d);

        hipMemcpy(gradA, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(gradUnified_d);
    }
    else
    {
        hipMemcpy(gradA, gradA_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradB_d, size * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipFree(gradA_d);
    hipFree(gradB_d);
    hipFree(gradResult_d);
}

float *mul_op(const int size, const float *valuesA, const float *valuesB)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *output = new float[size];

    float *valuesA_d;
    float *valuesB_d;
    float *output_d;

    hipMalloc((void **)&valuesA_d, size * sizeof(float));
    hipMalloc((void **)&valuesB_d, size * sizeof(float));
    hipMalloc((void **)&output_d, size * sizeof(float));

    hipMemcpy(valuesA_d, valuesA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesB_d, valuesB, size * sizeof(float), hipMemcpyHostToDevice);

    mulKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, valuesA_d, valuesB_d, output_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(output, output_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(valuesA_d);
    hipFree(valuesB_d);
    hipFree(output_d);

    return output;
}

void mulBackprop_op(const int size, float *gradA, float *gradB, const float *gradResult, const float *valuesA, const float *valuesB)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *gradA_d;
    float *gradB_d;
    float *gradResult_d;
    float *valuesA_d;
    float *valuesB_d;

    hipMalloc((void **)&gradA_d, size * sizeof(float));
    hipMalloc((void **)&gradB_d, size * sizeof(float));
    hipMalloc((void **)&gradResult_d, size * sizeof(float));
    hipMalloc((void **)&valuesA_d, size * sizeof(float));
    hipMalloc((void **)&valuesB_d, size * sizeof(float));

    hipMemcpy(gradA_d, gradA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradB_d, gradB, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradResult_d, gradResult, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesA_d, valuesA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesB_d, valuesB, size * sizeof(float), hipMemcpyHostToDevice);

    mulBackpropKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradResult_d, valuesA_d, valuesB_d);

    if (gradA == gradB)
    { // if same reference then we need to add them
        float *gradUnified_d;
        hipMalloc((void **)&gradUnified_d, size * sizeof(float));
        addKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradUnified_d);

        hipMemcpy(gradA, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(gradUnified_d);
    }
    else
    {
        hipMemcpy(gradA, gradA_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradB_d, size * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipFree(gradA_d);
    hipFree(gradB_d);
    hipFree(gradResult_d);
    hipFree(valuesA_d);
    hipFree(valuesB_d);
}

float *div_op(const int size, const float *valuesA, const float *valuesB)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *output = new float[size];

    float *valuesA_d;
    float *valuesB_d;
    float *output_d;

    hipMalloc((void **)&valuesA_d, size * sizeof(float));
    hipMalloc((void **)&valuesB_d, size * sizeof(float));
    hipMalloc((void **)&output_d, size * sizeof(float));

    hipMemcpy(valuesA_d, valuesA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesB_d, valuesB, size * sizeof(float), hipMemcpyHostToDevice);

    divKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, valuesA_d, valuesB_d, output_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(output, output_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(valuesA_d);
    hipFree(valuesB_d);
    hipFree(output_d);

    return output;
}

void divBackprop_op(const int size, float *gradA, float *gradB, const float *gradResult, const float *valuesA, const float *valuesB)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *gradA_d;
    float *gradB_d;
    float *gradResult_d;
    float *valuesA_d;
    float *valuesB_d;

    hipMalloc((void **)&gradA_d, size * sizeof(float));
    hipMalloc((void **)&gradB_d, size * sizeof(float));
    hipMalloc((void **)&gradResult_d, size * sizeof(float));
    hipMalloc((void **)&valuesA_d, size * sizeof(float));
    hipMalloc((void **)&valuesB_d, size * sizeof(float));

    hipMemcpy(gradA_d, gradA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradB_d, gradB, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradResult_d, gradResult, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesA_d, valuesA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesB_d, valuesB, size * sizeof(float), hipMemcpyHostToDevice);

    divBackpropKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradResult_d, valuesA_d, valuesB_d);

    if (gradA == gradB)
    { // if same reference then we need to add them

        float *ggA = (float*)malloc(size * sizeof(float));
        float *ggB = (float *)malloc(size * sizeof(float));
        hipMemcpy(ggA, gradA_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(ggB, gradB_d, size * sizeof(float), hipMemcpyDeviceToHost);

        float *gradUnified_d;
        hipMalloc((void **)&gradUnified_d, size * sizeof(float));
        addKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradUnified_d);

        hipMemcpy(gradA, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(gradUnified_d);
    }
    else
    {
        hipMemcpy(gradA, gradA_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradB_d, size * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipFree(gradA_d);
    hipFree(gradB_d);
    hipFree(gradResult_d);
    hipFree(valuesA_d);
    hipFree(valuesB_d);
}

float *pow_op(const int size, const float *valuesA, const float *valuesB)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *output = new float[size];

    float *valuesA_d;
    float *valuesB_d;
    float *output_d;

    hipMalloc((void **)&valuesA_d, size * sizeof(float));
    hipMalloc((void **)&valuesB_d, size * sizeof(float));
    hipMalloc((void **)&output_d, size * sizeof(float));

    hipMemcpy(valuesA_d, valuesA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesB_d, valuesB, size * sizeof(float), hipMemcpyHostToDevice);

    powKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, valuesA_d, valuesB_d, output_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(output, output_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(valuesA_d);
    hipFree(valuesB_d);
    hipFree(output_d);

    return output;
}

void powBackprop_op(const int size, float *gradA, float *gradB, const float *gradResult, const float *valuesA, const float *valuesB)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *gradA_d;
    float *gradB_d;
    float *gradResult_d;
    float *valuesA_d;
    float *valuesB_d;

    hipMalloc((void **)&gradA_d, size * sizeof(float));
    hipMalloc((void **)&gradB_d, size * sizeof(float));
    hipMalloc((void **)&gradResult_d, size * sizeof(float));
    hipMalloc((void **)&valuesA_d, size * sizeof(float));
    hipMalloc((void **)&valuesB_d, size * sizeof(float));

    hipMemcpy(gradA_d, gradA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradB_d, gradB, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradResult_d, gradResult, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesA_d, valuesA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(valuesB_d, valuesB, size * sizeof(float), hipMemcpyHostToDevice);

    powBackpropKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradResult_d, valuesA_d, valuesB_d);

    if (gradA == gradB)
    { // if same reference then we need to add them
        float *gradUnified_d;
        hipMalloc((void **)&gradUnified_d, size * sizeof(float));
        addKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, gradA_d, gradB_d, gradUnified_d);

        hipMemcpy(gradA, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradUnified_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(gradUnified_d);
    }
    else
    {
        hipMemcpy(gradA, gradA_d, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradB, gradB_d, size * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipFree(gradA_d);
    hipFree(gradB_d);
    hipFree(gradResult_d);
    hipFree(valuesA_d);
    hipFree(valuesB_d);
}

//Unary Ops
float* neg_op(const int size, const float *values){
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *output = new float[size];

    float *values_d;
    float *output_d;

    hipMalloc((void **)&values_d, size * sizeof(float));
    hipMalloc((void **)&output_d, size * sizeof(float));

    hipMemcpy(values_d, values, size * sizeof(float), hipMemcpyHostToDevice);

    negKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, values_d, output_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(output, output_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(values_d);
    hipFree(output_d);

    return output;
}
void negBackprop_op(const int size, float *grad, const float *gradResult)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *grad_d;
    float *gradResult_d;

    hipMalloc((void **)&grad_d, size * sizeof(float));
    hipMalloc((void **)&gradResult_d, size * sizeof(float));

    hipMemcpy(grad_d, grad, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradResult_d, gradResult, size * sizeof(float), hipMemcpyHostToDevice);

    negBackpropKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, grad_d, gradResult_d);

    hipMemcpy(grad, grad_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipFree(grad_d);
    hipFree(gradResult_d);
}
float *exp_op(const int size, const float *values)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *output = new float[size];

    float *values_d;
    float *output_d;

    hipMalloc((void **)&values_d, size * sizeof(float));
    hipMalloc((void **)&output_d, size * sizeof(float));

    hipMemcpy(values_d, values, size * sizeof(float), hipMemcpyHostToDevice);

    expKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, values_d, output_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(output, output_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(values_d);
    hipFree(output_d);

    return output;
}
void expBackprop_op(const int size, float *grad, const float *gradResult, const float *values)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *grad_d;
    float *gradResult_d;
    float *values_d;

    hipMalloc((void **)&grad_d, size * sizeof(float));
    hipMalloc((void **)&gradResult_d, size * sizeof(float));
    hipMalloc((void **)&values_d, size * sizeof(float));

    hipMemcpy(grad_d, grad, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradResult_d, gradResult, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(values_d, values, size * sizeof(float), hipMemcpyHostToDevice);

    expBackpropKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, grad_d, gradResult_d, values_d);

    hipMemcpy(grad, grad_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipFree(grad_d);
    hipFree(gradResult_d);
    hipFree(values_d);
}
float *tanh_op(const int size, const float *values)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *output = new float[size];

    float *values_d;
    float *output_d;

    hipMalloc((void **)&values_d, size * sizeof(float));
    hipMalloc((void **)&output_d, size * sizeof(float));

    hipMemcpy(values_d, values, size * sizeof(float), hipMemcpyHostToDevice);

    tanhKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, values_d, output_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(output, output_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(values_d);
    hipFree(output_d);

    return output;
}
void tanhBackprop_op(const int size, float *grad, const float *gradResult, const float *values)
{
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *grad_d;
    float *gradResult_d;
    float *values_d;

    hipMalloc((void **)&grad_d, size * sizeof(float));
    hipMalloc((void **)&gradResult_d, size * sizeof(float));
    hipMalloc((void **)&values_d, size * sizeof(float));

    hipMemcpy(grad_d, grad, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradResult_d, gradResult, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(values_d, values, size * sizeof(float), hipMemcpyHostToDevice);

    tanhBackpropKernel<<<GRID_SIZE, BLOCK_SIZE>>>(size, grad_d, gradResult_d, values_d);

    hipMemcpy(grad, grad_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipFree(grad_d);
    hipFree(gradResult_d);
    hipFree(values_d);
}

//Reduction Ops

float* sum_op(const int* shape, const int shapeSize, const int dimToReduce, const float* values)
{
    int BLOCK_SIZE = 32;

    int size = 1;
    for (int i = 0; i < shapeSize; i++)
    {
        size *= shape[i];
    }
    int outSize = size / shape[dimToReduce];

    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float* output = new float[outSize];

    float* values_d;
    float* output_d;
    int* shape_d;

    hipMalloc((void**)&values_d, size * sizeof(float));
    hipMalloc((void**)&output_d, outSize * sizeof(float));
    hipMalloc((void **)&shape_d, shapeSize * sizeof(int));

    hipMemcpy(values_d, values, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(shape_d, shape, shapeSize * sizeof(int), hipMemcpyHostToDevice);

    sumKernel<<<GRID_SIZE, BLOCK_SIZE >>> (shape_d, shapeSize, dimToReduce, values_d, output_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(output, output_d, outSize * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(values_d);
    hipFree(output_d);
    hipFree(shape_d);

    return output;
}

void sumBackprop_op(const int *shape, const int shapeSize, const int dimToReduce, float *grad, const float *gradResult)
{
    int BLOCK_SIZE = 32;

    int size = 1;
    for (int i = 0; i < shapeSize; i++)
    {
        size *= shape[i];
    }
    int outSize = size / shape[dimToReduce];

    int GRID_SIZE = (int)ceil(size / (float)BLOCK_SIZE);

    float *grad_d;
    float *gradResult_d;
    int *shape_d;

    hipMalloc((void **)&grad_d, size * sizeof(float));
    hipMalloc((void **)&gradResult_d, size * sizeof(float));
    hipMalloc((void **)&shape_d, shapeSize * sizeof(int));

    hipMemcpy(grad_d, grad, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gradResult_d, gradResult, outSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(shape_d, shape, shapeSize * sizeof(int), hipMemcpyHostToDevice);

    sumBackpropKernel<<<GRID_SIZE, BLOCK_SIZE>>>(shape_d, shapeSize, dimToReduce, grad_d, gradResult_d);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(grad, grad_d, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(grad_d);
    hipFree(gradResult_d);
    hipFree(shape_d);
}